
#include <hip/hip_runtime.h>
extern "C" {

// CUDA核函数（CPU主机端调用，GPU设备端执行）
// CUDA核函数标识符：__global__
// CUDA核函数返回值类型必须是void
__global__ void hello_cuda_from_gpu(int n) {
    printf("GPU: 你好, CUDA! (Rust版)\n");
}

}